#include <cuda/common.cuh>

#include <hip/hip_runtime.h>

namespace vo {
    namespace cuda {

        void cuda_init_device(){
            // NOTE: Not support for multiple GPUs yet
            vo::cuda::query_devices();
            HANDLE_CUDA_ERROR(hipSetDeviceFlags(hipDeviceMapHost));
            HANDLE_CUDA_ERROR(hipSetDevice(0));
        }

        void handle_cuda_error(int err, const char* file, int line)
        {
            if(err != hipSuccess)
            {
                printf("%s in %s at line %d\n", hipGetErrorString((hipError_t) err), file, line);
                exit(EXIT_FAILURE);
            }
        };
        
        void query_devices()
        {
            int dev_count;
            hipGetDeviceCount(&dev_count);
            for(int i = 0; i < dev_count; i++)
            {
                hipDeviceProp_t dev_prop;
                hipGetDeviceProperties(&dev_prop, i);
                printf("Found CUDA Capable device %s (%d.%d)\n", dev_prop.name, 
                                                            dev_prop.major,
                                                            dev_prop.minor);
            }
        }

        void cuda_malloc_wrapper(void **devPtr, size_t size) {
            HANDLE_CUDA_ERROR(hipMalloc(devPtr, size));
        }

        void cuda_memcpy_to_device_wrapper(void *dst, const void *src, size_t count) {
            HANDLE_CUDA_ERROR(hipMemcpy(dst, src, count, hipMemcpyKind::hipMemcpyHostToDevice));
        }

        void cuda_memcpy_to_host_wrapper(void *dst, const void *src, size_t count) {
            HANDLE_CUDA_ERROR(hipMemcpy(dst, src, count, hipMemcpyKind::hipMemcpyDeviceToHost));
        }

        void cuda_malloc_managed_wrapper(void **devPtr, size_t size, unsigned int flags) {
            HANDLE_CUDA_ERROR(hipMallocManaged(devPtr, size, flags));
        }

        void cuda_free_wrapper(void *devPtr) {
            HANDLE_CUDA_ERROR(hipFree(devPtr));
        }

    } // namespace cuda
} // namespace vo